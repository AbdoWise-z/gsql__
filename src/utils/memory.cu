//
// Created by xabdomo on 4/13/25.
//

#include "memory.cuh"


void * cu::malloc(size_t size) {
    void* ptr;
    hipMalloc(&ptr, size);
    return ptr;
}

void cu::free(void *ptr) {
    hipFree(ptr);
}

void cu::toDevice(void *src, void *dst, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void cu::toHost(void *src, void *dst, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}
