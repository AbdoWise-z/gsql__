//
// Created by xabdomo on 4/13/25.
//

#include "memory.cuh"

#include <stdexcept>


void * cu::malloc(size_t size) {
    void* ptr;
    auto err = hipMalloc(&ptr, size);
    if (err != hipSuccess) {
        throw std::runtime_error("CUDA malloc failed: " + std::string(hipGetErrorString(err)));
    }
    return ptr;
}

void cu::free(void *ptr) {
    hipFree(ptr);
}

void cu::toDevice(void *src, void *dst, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void cu::toHost(void *src, void *dst, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}
