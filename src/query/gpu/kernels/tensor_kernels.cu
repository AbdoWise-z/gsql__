#include "hip/hip_runtime.h"
//
// Created by xabdomo on 4/20/25.
//

#include "tensor_kernels.cuh"
#include "constants.hpp"
#include "utils/murmur_hash3_cuda.cuh"

__global__ void TensorKernel::fill_kernel(char *output_data, char value, size_t size)  {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output_data[idx] = value;
    }
}

__global__ void TensorKernel::fill_kernel(char *output_data, size_t dataSize, char value, size_t* center_pos, size_t *mask, size_t *shape, size_t maskSize) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= dataSize) {
        return;
    }

    size_t pos[MAX_TENSOR_DIMS];
    unmap(shape, pos, idx, maskSize);

    bool _set = true;
    for (int i = 0; i < maskSize; ++i) {
        if (!mask[i] && pos[i] != center_pos[i]) {
            _set = false;
            break;
        }
    }

    if (_set)
        output_data[idx] = value;
}

__device__ __host__ void TensorKernel::unmap(size_t *shape, size_t *pos, size_t index, size_t size) {
    size_t remaining = index;
    for (int i = 0; i < size; ++i) {
        pos[i] = remaining % shape[i];
        remaining /= shape[i];
    }
}

__device__ __host__ size_t TensorKernel::map(size_t *indices, size_t *shape, size_t size) {
    size_t index = 0;
    size_t acc = 1;
    for (size_t i = 0;i < size;i++) {
        index += indices[i] * acc;
        acc *= shape[i];
    }
    return index;
}

__global__ void TensorKernel::extend_plain_kernel(
        char *output_data,
        size_t dataSize,
        size_t *mask,
        size_t *shape,
        size_t maskSize
    ) {

    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= dataSize) {
        return;
    }

    size_t pos[MAX_TENSOR_DIMS];
    unmap(shape, pos, idx, maskSize);

    for (int i = 0; i < maskSize; ++i) {
        if (mask[i]) {
            pos[i] = 0; // load the data from the zero-th plain
            break;
        }
    }

    output_data[idx] = output_data[map(pos, shape, maskSize)];
}

__global__ void TensorKernel::logical_and(char *a, char *b, size_t size, char *out) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = a[idx] && b[idx];
    }
}

__global__ void TensorKernel::logical_or(char *a, char *b, size_t size, char *out) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = a[idx] || b[idx];
    }
}

__global__ void TensorKernel::logical_not(char *a, size_t size, char *out) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = !a[idx];
    }
}


