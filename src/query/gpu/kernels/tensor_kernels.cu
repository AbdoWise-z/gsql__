#include "hip/hip_runtime.h"
//
// Created by xabdomo on 4/20/25.
//

#include "tensor_kernels.cuh"
#include "constants.hpp"
#include "utils/murmur_hash3_cuda.cuh"

__global__ void TensorKernel::fill_kernel(char *output_data, char value, size_t size)  {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output_data[idx] = value;
    }
}

__global__ void TensorKernel::fill_kernel(char *output_data, size_t dataSize, char value, size_t* center_pos, size_t *mask, size_t *shape, size_t maskSize) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= dataSize) {
        return;
    }

    size_t pos[MAX_TENSOR_DIMS];
    unmap(shape, pos, idx, maskSize);

    bool _set = true;
    for (int i = 0; i < maskSize; ++i) {
        if (!mask[i] && pos[i] != center_pos[i]) {
            _set = false;
            break;
        }
    }

    if (_set)
        output_data[idx] = value;
}

__device__ void TensorKernel::unmap(size_t *shape, size_t *pos, size_t index, size_t size) {
    size_t remaining = index;
    for (int i = 0; i < size; ++i) {
        pos[i] = remaining % shape[i];
        remaining /= shape[i];
    }
}

size_t TensorKernel::map(size_t *indices, size_t *shape, size_t size) {
    size_t index = 0;
    size_t acc = 1;
    for (size_t i = 0;i < size;i++) {
        index += indices[i] * acc;
        acc *= shape[i];
    }
    return index;
}

__global__ void TensorKernel::extend_plain_kernel(
        char *output_data,
        size_t dataSize,
        size_t *mask,
        size_t *shape,
        size_t maskSize
    ) {

    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= dataSize) {
        return;
    }

    size_t pos[MAX_TENSOR_DIMS];
    unmap(shape, pos, idx, maskSize);

    for (int i = 0; i < maskSize; ++i) {
        if (mask[i]) {
            pos[i] = 0; // load the data from the zero-th plain
            break;
        }
    }

    output_data[idx] = output_data[map(pos, shape, maskSize)];
}
